#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA kernel to transform elements
__global__ void multiply10(float *data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = data[idx] * 10.0f;
    }
}

int main()
{
    constexpr uint64_t num_elements = 1'000'000'000;
    const size_t bytes = num_elements * sizeof(float);

    std::cout << "Allocating " << num_elements << " float elements (~4GB)..." << std::endl;
    
    // Host data
    float *h_data = new float[num_elements];
    for (uint64_t i = 0; i < num_elements; i++)
    {
        h_data[i] = 1.0f;
    }
    
    // Device data
    float *d_data = nullptr;
    hipMalloc(&d_data, bytes);
    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
    
    // Set up CUDA kernel execution
    const int blockSize = 256;
    const int gridSize = (num_elements + blockSize - 1) / blockSize;
    
    const auto start = std::chrono::high_resolution_clock::now();
    
    // Launch kernel
    multiply10<<<gridSize, blockSize>>>(d_data, num_elements);
    
    // Wait for GPU to finish
    hipDeviceSynchronize();

    const auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_time = end - start;
    
    // Copy result back to host
    hipMemcpy(h_data, d_data, bytes, hipMemcpyDeviceToHost);

    std::cout << "First and last element: " << h_data[0] << " " << h_data[num_elements - 1] << std::endl;
    std::cout << "Computation completed in " << total_time.count() << " seconds" << std::endl;
    std::cout << "Total time: " << total_time.count() << " seconds" << std::endl;
    
    // Clean up
    hipFree(d_data);
    delete[] h_data;
    
    return 0;
}