#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION  
#include "stb_image_write.h"
#include <iostream>
#include <vector>

__global__ void colorToGrayscaleKernel(const uint8_t* colorInput, uint8_t* grayOutput)
{
    const int x = threadIdx.x;  // 0 ~ 31
    const int y = threadIdx.y;  // 0 ~ 31
    
    const int colorIdx = (y * 32 + x) * 3;

    const int grayIdx = y * 32 + x;

    const float r = static_cast<float>(colorInput[colorIdx + 0]);
    const float g = static_cast<float>(colorInput[colorIdx + 1]);
    const float b = static_cast<float>(colorInput[colorIdx + 2]);

    const float gray = 0.299f * r + 0.587f * g + 0.114f * b;

    grayOutput[grayIdx] = static_cast<uint8_t>(gray);
}

int main()
{
    // 32×32 컬러 이미지 로드 (3채널)
    int imgWidth, imgHeight, imgChannels;
    uint8_t* hostColorImage = stbi_load("cat32color.png",
        &imgWidth, &imgHeight, &imgChannels, 3); 
    
    assert(imgWidth == 32 && imgHeight == 32 && imgChannels == 3);
    
    uint8_t* deviceColorInput;
    uint8_t* deviceGrayOutput;
    hipMalloc(&deviceColorInput, 32 * 32 * 3 * sizeof(uint8_t)); 
    hipMalloc(&deviceGrayOutput, 32 * 32 * sizeof(uint8_t));     
    
    hipMemcpy(deviceColorInput, hostColorImage, 32 * 32 * 3, hipMemcpyHostToDevice);
    
    constexpr dim3 blockSize(32, 32); 
    colorToGrayscaleKernel<<<1, blockSize>>>(deviceColorInput, deviceGrayOutput);
    
    hipDeviceSynchronize();
    
    // 흑백 결과를 CPU로 복사
    std::vector<uint8_t> hostGrayResult(32 * 32);
    hipMemcpy(hostGrayResult.data(), deviceGrayOutput, 32 * 32, hipMemcpyDeviceToHost);
    
    // 흑백 이미지 저장
    stbi_write_png("cat32gray_converted.png", 32, 32, 1, hostGrayResult.data(), 32);
    
    // 메모리 해제
    hipFree(deviceColorInput);
    hipFree(deviceGrayOutput);
    stbi_image_free(hostColorImage);
    
    std::cout << "Color to grayscale conversion completed!" << std::endl;
    return 0;
}