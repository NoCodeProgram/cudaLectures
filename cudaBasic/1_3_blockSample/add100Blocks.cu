#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void add100(int32_t* data) 
{
    const int idx = threadIdx.x;
    data[idx] = data[idx] + 100;
}

int main()
{
    constexpr uint32_t dataLength = 1024 * 100;
    std::vector<int32_t> hostData(dataLength);
    
    for (uint32_t i = 0; i < dataLength; ++i)
    {
        hostData[i] = static_cast<int32_t>(i);
    }

    int32_t* deviceData;
    hipMalloc(&deviceData, dataLength * sizeof(int32_t));    
    hipMemcpy(deviceData, hostData.data(), dataLength * sizeof(int32_t), hipMemcpyHostToDevice);
    
    constexpr int32_t blockSize = 1024;
    constexpr int32_t numBlocks = 100;
    add100<<<numBlocks, blockSize>>>(deviceData);

    hipDeviceSynchronize();
    hipMemcpy(hostData.data(), deviceData, dataLength * sizeof(int32_t), hipMemcpyDeviceToHost);
    
    hipFree(deviceData);

    for(uint32_t idx = datalength - 10; idx < dataLength; ++idx)
    {
        std::cout << hostData[idx] << " ";
    }
    std::cout << std::endl;
    return 0;
}