#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <source_location>

inline void cudaCheckErr(hipError_t err, const std::source_location& loc = std::source_location::current())
{
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << loc.file_name() << ":" << loc.line() 
                  << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void add100(int32_t* data)
{
    const int idx = threadIdx.x;
    data[idx] = data[idx] + 100;
}

int main()
{
    constexpr uint32_t dataLength = 1025;
    std::vector<int32_t> hostData(dataLength);
    for (int32_t i = 0; i < dataLength; ++i)
    {
        hostData[i] = i; 
    }

    int32_t* deviceData;
    const auto mallocErr = hipMalloc(&deviceData, dataLength * sizeof(int32_t));
    cudaCheckErr(mallocErr);

    cudaCheckErr(hipMemcpy(deviceData, hostData.data(), dataLength * sizeof(int32_t), hipMemcpyHostToDevice));
    add100 <<<1, dataLength >>> (deviceData);
    const hipError_t launchErr = hipGetLastError();    
    cudaCheckErr(launchErr);

    cudaCheckErr(hipDeviceSynchronize());

    cudaCheckErr(hipMemcpy(hostData.data(), deviceData, dataLength * sizeof(int32_t), hipMemcpyDeviceToHost));
    hipFree(deviceData);

    for (int32_t i = 0; i < 10; ++i)
    {
        std::cout << hostData[i] << " ";
    }    
    return 0;
}