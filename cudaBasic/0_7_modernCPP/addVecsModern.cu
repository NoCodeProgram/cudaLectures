
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int32_t* dataA, const int32_t* dataB, int32_t* dataC)
{
    const int idx = threadIdx.x;
    dataC[idx] = dataA[idx] + dataB[idx];
}

int main()
{
    constexpr uint32_t dataLength = 1024;

    std::vector<int32_t> hostDataA(dataLength);
    std::vector<int32_t> hostDataB(dataLength);
    std::vector<int32_t> hostDataC(dataLength);

// Initialize data
    for (int32_t i = 0; i < dataLength; ++i)
    {
        hostDataA[i] = i;// A = [0, 1, 2, 3, ...]
        hostDataB[i] = i * 2;// B = [0, 2, 4, 6, ...]
        hostDataC[i] = 0;// C = [0, 0, 0, 0, ...]
    }

// Allocate device memory
    int32_t* deviceDataA = nullptr;
    int32_t* deviceDataB = nullptr;
    int32_t* deviceDataC = nullptr;

    hipMalloc(&deviceDataA, dataLength * sizeof(int32_t));
    hipMalloc(&deviceDataB, dataLength * sizeof(int32_t));
    hipMalloc(&deviceDataC, dataLength * sizeof(int32_t));

// Copy host to device memory
    hipMemcpy(deviceDataA, hostDataA.data(), dataLength * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceDataB, hostDataB.data(), dataLength * sizeof(int32_t), hipMemcpyHostToDevice);

// Launch kernel
    vectorAdd <<<1, dataLength >>> (deviceDataA, deviceDataB, deviceDataC);

// Synchronize
    hipDeviceSynchronize();

// Copy device to host memory
    hipMemcpy(hostDataC.data(), deviceDataC, dataLength * sizeof(int32_t), hipMemcpyDeviceToHost);

// Print results (first 10 and last 10 elements)
    std::cout << "First 10 : ";
    for (int32_t i = 0; i < 10; ++i)
    {
        std::cout << hostDataC[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Last 10 : ";
    for (int32_t i = dataLength - 10; i < static_cast<int32_t>(dataLength); ++i)
    {
        std::cout << hostDataC[i] << " ";
    }
    std::cout << std::endl;

// Free memory
    hipFree(deviceDataA);
    hipFree(deviceDataB);
    hipFree(deviceDataC);

    return 0;
}
