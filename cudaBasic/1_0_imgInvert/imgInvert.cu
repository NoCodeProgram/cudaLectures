
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int32_t* dataA, const int32_t* dataB, int32_t* dataC)
{
    const int idx = threadIdx.x;
    dataC[idx] = dataA[idx] + dataB[idx];
}

void cpuVectorAdd(const int32_t* dataA, const int32_t* dataB, int32_t* dataC, const int size)
{
    for(int32_t idx = 0; idx < size; ++idx)
    {
        dataC[idx] = dataA[idx] + dataB[idx];
    }
}

int main()
{
    constexpr uint32_t dataLength = 1024;

// Allocate host memory
    int32_t *hostDataA = new int32_t[dataLength];
    int32_t *hostDataB = new int32_t[dataLength];
    int32_t *hostDataC = new int32_t[dataLength];

// Initialize data
    for (int32_t i = 0; i < dataLength; ++i)
    {
        hostDataA[i] = i;// A = [0, 1, 2, 3, ...]
        hostDataB[i] = i * 2;// B = [0, 2, 4, 6, ...]
        hostDataC[i] = 0;// C = [0, 0, 0, 0, ...]
    }

// Allocate device memory
    int32_t* deviceDataA = nullptr;
    int32_t* deviceDataB = nullptr;
    int32_t* deviceDataC = nullptr;

    hipMalloc(&deviceDataA, dataLength * sizeof(int32_t));
    hipMalloc(&deviceDataB, dataLength * sizeof(int32_t));
    hipMalloc(&deviceDataC, dataLength * sizeof(int32_t));

// Copy host to device memory
    hipMemcpy(deviceDataA, hostDataA, dataLength * sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(deviceDataB, hostDataB, dataLength * sizeof(int32_t), hipMemcpyHostToDevice);

// Launch kernel
    vectorAdd <<<1, dataLength >>> (deviceDataA, deviceDataB, deviceDataC);

// Synchronize
    hipDeviceSynchronize();

// Copy device to host memory
    hipMemcpy(hostDataC, deviceDataC, dataLength * sizeof(int32_t), hipMemcpyDeviceToHost);

// Print results (first 10 and last 10 elements)
    std::cout << "First 10 : ";
    for (int32_t i = 0; i < 10; ++i)
    {
        std::cout << hostDataC[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Last 10 : ";
    for (int32_t i = dataLength - 10; i < static_cast<int32_t>(dataLength); ++i)
    {
        std::cout << hostDataC[i] << " ";
    }
    std::cout << std::endl;

// Free memory
    hipFree(deviceDataA);
    hipFree(deviceDataB);
    hipFree(deviceDataC);

    delete[] hostDataA;
    delete[] hostDataB;
    delete[] hostDataC;

    return 0;
}
